
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_vector_types.h>

typedef unsigned char   uint8;
typedef unsigned int    uint32;
typedef int             int32;

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

__constant__ uint32 constAlpha;

#define MUL(x,y)    (x*y)
__constant__ float  HueColorSpaceAry[9];


__device__ void YUV2RGB(uint32 *yuv, float *red, float *green, float *blue)
{
    float luma, chromaCb, chromaCr;

    // Prepare for hue adjustment
    luma     = (float)yuv[0];
    chromaCb = (float)((int32)yuv[1] - 512.0f);
    chromaCr = (float)((int32)yuv[2] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    *red  = MUL(luma,     HueColorSpaceAry[0]) +
            MUL(chromaCb, HueColorSpaceAry[1]) +
            MUL(chromaCr, HueColorSpaceAry[2]);
    *green= MUL(luma,     HueColorSpaceAry[3]) +
            MUL(chromaCb, HueColorSpaceAry[4]) +
            MUL(chromaCr, HueColorSpaceAry[5]);
    *blue = MUL(luma,     HueColorSpaceAry[6]) +
            MUL(chromaCb, HueColorSpaceAry[7]) +
            MUL(chromaCr, HueColorSpaceAry[8]);
}


__device__ uint32 ARGB_10bit(float red, float green, float blue, uint32 alpha)
{
    uint32 ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 1023.f);
    green = min(max(green, 0.0f), 1023.f);
    blue  = min(max(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = (((uint32)blue  >> 2) |
                 (((uint32)green >> 2) << 8)  |
                 (((uint32)red   >> 2) << 16) | (uint32)alpha);

    return  ARGBpixel;
}


// CUDA kernel for outputing the final ARGB output from NV12;
extern "C"
__global__ void NV12ToARGB(uint32 *srcImage,     size_t nSourcePitch,
                           uint32 *dstImage,     size_t nDestPitch,
                           uint32 width,         uint32 height)
{
    int32 x, y;
    uint32 yuv101010Pel[2];
    uint32 processingPitch = ((width) + 63) & ~63;
    uint32 dstImagePitch   = nDestPitch >> 2;
    uint8 *srcImageU8      = (uint8 *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return;

    if (y >= height)
        return;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32 chromaOffset    = processingPitch * height;
    int32 y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32 chromaCb;
        uint32 chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((uint32)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    uint32 yuv[6];
    float red[2], green[2], blue[2];

    yuv[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
    yuv[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuv[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuv[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuv[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuv[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuv[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuv[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x     ] = ARGB_10bit(red[0], green[0], blue[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1 ] = ARGB_10bit(red[1], green[1], blue[1], constAlpha);
}